#include "hip/hip_runtime.h"
#include <stdlib.h>
#include <stdio.h>
#include <hip/hip_runtime.h>

#include "common.h"

__global__ void kernel_sqrt(double* input_arr, double* output_arr, size_t len) {
    unsigned int tid = blockIdx.x * blockDim.x + threadIdx.x;
    if(tid < len) {
        output_arr[tid] = sqrt(input_arr[tid]);
    }
}

void get_sqrt(double* arr, size_t len) {
    size_t nbytes = len * sizeof(double);
    double* d_arr = NULL;
    double* d_ret = NULL;
    CUDAErrorCheck(hipMalloc((void**)&d_arr, nbytes));
    CUDAErrorCheck(hipMalloc((void**)&d_ret, nbytes));
    CUDAErrorCheck(hipMemcpy(d_arr, arr, nbytes, hipMemcpyHostToDevice));
    kernel_sqrt<<<GPU_BLOCKS_PER_GRID, GPU_THREADS_PER_BLOCK>>>(d_arr, d_ret, len);
    CUDAErrorCheck(hipMemcpy(arr, d_ret, nbytes, hipMemcpyDeviceToHost)); // inplace sqrt of array
    CUDAErrorCheck(hipFree(d_arr));
    CUDAErrorCheck(hipFree(d_ret));
} 
